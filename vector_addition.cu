
#include <hip/hip_runtime.h>
# include <iostream>
__global__ void vector_add(const float* A, const float* B,  float* C, int N){
    int i = blockIdx.x * blockDim.x +  threadIdx.x;
    if (i < N){
        C[i] = A[i] + B[i];
    }
}

int main(){
    const int N = 10;
    float A[N], B[N], C[N];

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N*sizeof(float));
    hipMalloc(&d_b, N*sizeof(float));
    hipMalloc(&d_c, N*sizeof(float));

    hipMemcpy(d_a, A,N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B,N*sizeof(float), hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = ceil(N/block_size);
    vector_add<<<grid_size, block_size >>>(d_a,d_b,d_c,N);

    hipMemcpy(C, d_c, N*sizeof(float),hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}